#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <math.h>
#include "caffe/layers/box_pooling_layer.hpp"

using std::max;
using std::min;

namespace caffe {

  template <typename Dtype>
  __global__ void BOXPoolingForward(
    const int nthreads,
    const int num_features, const int num_regions,const int num_samples,
    const Dtype* bottom_data,
    const int height, const int width,
    Dtype* top_data,
    const int num_rfcn_regions,
    int* mapping_channel) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The input is in order (1, nr*nf , H, W), nf = 64
      
      int output_dim = num_features * num_regions;
      int w = index % width;
      int h = (index / width) % height;
      int ctop = (index / width / height) % output_dim; // outdim = N_feature* n_region
      int feature = (index / width / height) % num_features;
      int region = (index / width / height/ num_features) % num_regions; 
      int n = index / width / height / output_dim;
      int direction = 4 * ctop / output_dim;
      int num_regions_one_side = (int)(sqrt(num_regions))
      int num_rfcn_regions_one_side = (int)(sqrt(num_rfcn_regions))
      int w_region = region% num_regions_one_side;
      int h_region = region/ num_regions_one_side;
      float centered_w_region = (float)w_region - (float)num_regions_one_side / 2.0)
      float centered_h_region = (float)h_region - (float)num_regions_one_side / 2.0)

      for (int i = 0;i< num_samples; i++){
        int x = sample_id[region*num_samples + i];
        int y = sample_id[num_regions* num_samples + region*num_samples + i];
        int sampled_x = w + x;
        int sampled_y = h + y;

        if (sampled_x >= 0 && sampled_x < width)&&(sampled_y >= 0 && sampled_y < height){
          
          int tmp[num_rfcn_regions] = {0};
          for (int j = 0; j< num_rfcn_region; j ++ ){
            
            int w_rfcn_region = j % num_rfcn_regions_one_side;
            int h_rfcn_region = j / num_rfcn_regions_one_side;
            float centered_w_rfcn_region = (float)w_rfcn_region - (float)num_rfcn_regions_one_side / 2.0)
            float centered_h_rfcn_region = (float)h_rfcn_region - (float)num_rfcn_regions_one_side / 2.0)
            if ((centered_w_rfcn_region* centered_w_region >0)&& (centered_h_rfcn_region* centered_h_region >0)){
              int bottom_idx = n*output_dim*height*width + j*num_features*height*width + sampled_y * width + sampled_x;
              for (int k = 0;k<num_features;k++){
                  tmp[j] += bottom_data[bottom_idx + k]*bottom_data[bottom_idx + k];
              }

            }
          }
          int selected = 0; int max_norm = 0;
          for (int j = 0; j< num_rfcn_region; j ++ ){
            if (tmp[j]> max_norm){
              max_norm = tmp[j];
              selected = j;
            }

          }
        
          int bottom_idx = n*output_dim*height*width + selected*num_features*height*width + feature * height*width + sampled_y * width + sampled_x;
          top_data[index] += bottom_data[bottom_idx];
          mapping_channel[ index*num_samples + i ] = index;
          mapping_channel[ nthreads*num_samples + index * num_samples + i ] = bottom_idx;
          // TODO: mapping back require search from the second half of mapping channels 

        }
        
      }
    }
  }

  template <typename Dtype>
  void BOXPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int* mapping_channel_ptr = mapping_channel_.mutable_gpu_data();
    int count = top[0]->count();
    caffe_gpu_set(count, Dtype(0), top_data);
    caffe_gpu_set(mapping_channel_.count(), -1, mapping_channel_ptr);
    // NOLINT_NEXT_LINE(whitespace/operators)
    BOXPoolingForward<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, num_features_, num_regions_,num_samples_,bottom_data, height_, width_,
      top_data, num_rfcn_regions_,mapping_channel_ptr);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  __global__ void BOXPoolingBackwardAtomic(
    const int nthreads,
    const Dtype* top_diff,
    const int* mapping_channel,
    const int num_features,
    const Dtype num_regions,
    const int num_samples,
    const int height, const int width,
    Dtype* bottom_diff,) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The output is in order (n, ctop, ph, pw)
      int output_dim = num_features * num_regions;
      int w = index % width;
      int h = (index / width) % height;
      int ctop = (index / width / height) % output_dim; // outdim = N_feature* n_region
      int feature = (index / width / height) % num_features;
      int region = (index / width / height/ num_features) % num_regions; 
      int n = index / width / height / output_dim;
      int direction = 4 * ctop / output_dim;
      
      for(int i = 0; i < nthreads * num_samples; i++ ){
        if ( mapping_channel[i + nthreads * num_samples ]== index){
          bottom_diff[ index ] += top_diff[mapping_channel[i]];
        } 
      }
    }
  }

  template <typename Dtype>
  void BOXPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
      return;
    }
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    const int* mapping_channel_ptr = mapping_channel_.gpu_data();
    caffe_gpu_set(bottom_count, Dtype(0), bottom_diff);
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    BOXPoolingBackwardAtomic<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, top_diff, mapping_channel_ptr,
      num_features_, num_regions_,num_samples_,  height_, width_, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(BOXPoolingLayer);

}  // namespace caffe
