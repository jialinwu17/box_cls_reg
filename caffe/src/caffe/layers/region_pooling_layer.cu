#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <math.h>
#include "caffe/layers/region_pooling_layer.hpp"

using std::max;
using std::min;

namespace caffe {

  template <typename Dtype>
  __global__ void BOXPoolingForward(
    const int nthreads,
    const int num_features, const int num_regions,const int num_samples, const int M, const Dtype * feat_ext_offset,
    const Dtype* bottom_data,
    const int height, const int width,
    Dtype* top_data,
    const int num_rfcn_regions,
    int* mapping_channel,
    const Dtype * seed_points,
    const Dtype* rfcn_regions_weights) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The input is in order (1, num_rfcn_regions * num_features , H, W ), nf = 64
      // seed_points is in order (m, 2)
      // feat_ext_offset is in order (m, num_regions, num_samples, 2)
      // rfcn_region_idx
      // output is in order (m, num_regions, num_samples, num_rfcn_regions, num_features)
      // nthreads m * num_regions * num_features

      for (int rfcn_r = 0 ; rfcn_r < num_rfcn_regions ; rfcn_r ++){
      int output_dim = num_features * num_regions;
      int f = index % num_features;
      int s = (index / num_features ) % num_samples;
      int r = (index / num_features / num_samples) % num_regions;
      int m = index / num_features  / num_samples / num_regions;
      int x = seed_points[ 2 * m ];
      int y = seed_points[ 2 * m + 1 ];
      int feat_ext_idx = feat_ext_offset [ m * num_regions * num_samples *2 +  2 * r * num_samples + 2 * s ] + x;
      int feat_ext_idy = feat_ext_offset [ m * num_regions * num_samples *2 +  2 * r * num_samples + 2 * s + 1] + y;
      int bottom_idx = ((rfcn_r * num_features + f) * height + feat_ext_idy) * width + feat_ext_idx;
      int map_channel_idx = ((f) * height + feat_ext_idy) * width + feat_ext_idx;
      int rfcn_regions_weights_idx = (((m * num_regions + r)* num_samples + s) * num_rfcn_regions + rfcn_r) ;
      top_data[index] += bottom_data[bottom_idx] * rfcn_regions_weights[rfcn_regions_weights_idx];
      mapping_channel[ index ] = map_channel_idx;
    }
    top_data[index] /=  num_rfcn_regions;
    }
  }

  template <typename Dtype>
  void RegionPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int* mapping_channel_ptr = mapping_channel_.mutable_gpu_data();
    int count = top[0]->count();
    caffe_gpu_set(count, Dtype(0), top_data);
    caffe_gpu_set(mapping_channel_.count(), -1, mapping_channel_ptr);
    // NOLINT_NEXT_LINE(whitespace/operators)
    BOXPoolingForward<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, num_features_, num_regions_,num_samples_, bottom[1]->shape(0), bottom[2]->gpu_data(), bottom_data, height_, width_,
      top_data, num_rfcn_regions_,mapping_channel_ptr,bottom[1]->gpu_data(),bottom[3]->gpu_data());
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  __global__ void BOXPoolingBackwardAtomic(
    const int nthreads,
    const Dtype* top_diff,
    const int* mapping_channel,
    const int num_features,
    const int num_regions,
    const int num_samples,
    const int height, const int width, const int M,
    Dtype* bottom_diff, const int num_rfcn_regions,const Dtype* rfcn_regions_weights) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      // The input is in order (1, num_rfcn_regions * num_features , H, W ), nf = 64
      // sampled_points is in order (m, 2)
      // feat_ext_offset is in order (num_regions, num_samples, 2)
      // rfcn_region_idx
      // output is in order (m, num_regions, num_samples, num_features)
      // nthreads m * num_regions * num_features

      int w = index % width;
      int h = (index / width) % height;
      int f = (index / width / height) % num_features;
      int rfcn_r = index / width / height / num_features;
      for (int i = 0; i < M * num_regions * num_samples; i++ ){
        int s = i % num_samples;
        int r = (i / num_samples) % num_regions;
        int m = i / num_samples / num_regions;
        int top_idx = (((m * num_regions + r)* num_samples + s))*num_features + f ;
        int rfcn_regions_weights_idx = (((m * num_regions + r)* num_samples + s) * num_rfcn_regions + rfcn_r) ;
        if (mapping_channel[top_idx] == ((f * height + h) * width + w)){
          bottom_diff[index] += top_diff[top_idx] *rfcn_regions_weights[rfcn_regions_weights_idx];
        }

      }
      bottom_diff[index] /= num_rfcn_regions;
    }
  }

  template <typename Dtype>
  void RegionPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
      return;
    }
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    const int* mapping_channel_ptr = mapping_channel_.gpu_data();
    caffe_gpu_set(bottom_count, Dtype(0), bottom_diff);
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    BOXPoolingBackwardAtomic<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, top_diff, mapping_channel_ptr,
      num_features_, num_regions_,num_samples_,  height_, width_, bottom[1]->shape(0), bottom_diff,num_rfcn_regions_,bottom[3]->gpu_data());
    CUDA_POST_KERNEL_CHECK;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(RegionPoolingLayer);

}  // namespace caffe
